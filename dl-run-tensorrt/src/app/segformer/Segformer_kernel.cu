#include "hip/hip_runtime.h"
#include "Segformer_kernel.h"
#include <opencv2/opencv.hpp>


static __global__ void process_image_kernel(uint8_t* input, float* output, int allPixel)
{
    int tid = blockDim.x * threadIdx.y + threadIdx.x;
    int blockNum = blockDim.x * blockDim.y;
    int block_offset = blockNum * blockIdx.x;
    int num_a_row = blockNum * gridDim.x;
    int row_offset = num_a_row * blockIdx.y;
    int position = tid + block_offset + row_offset;

    if (position >= allPixel) 
    {
        return;
    }
    //RGB -> RRRRRRRRR;GGGGGGGG;BBBBBBBB
    int inputIndex = position * 3;
    output[position] = input[inputIndex] / 255.0f;
    output[position + allPixel] = input[inputIndex + 1] / 255.0f;
    output[position + allPixel * 2] = input[inputIndex + 2] / 255.0f;
    //printf("input %d  ,%d  , %d\n", p1, *p2, *p3);
    //printf("output %f  ,%f  , %f\n", output[index], output[index+1], output[index+2]);
    //uint8_t* p1 = input;  // ָ����������ĵ�һ��Ԫ��
    //printf("First channel value in the input: %d\n", position);
}



void app::process_image(uint8_t* input, float* output, int height, int width, hipStream_t stream)
{

    // mask_weights is mask_dim(32 element) gpu pointer
    dim3 grid((width + 31) / 32, (height + 31) / 32);
    dim3 block(32, 32);

    process_image_kernel << <grid, block, 0, stream >> > (input, output,height* width);
}

static __global__ void post_process_result_kernel(uint32_t* input, uint8_t* output, int count) 
{
    int position = (blockDim.x * blockIdx.x + threadIdx.x);
    if (position>count)
    {
        return;
    }
    output[position] = input[position];
}
void app::post_process_result(uint32_t* input, uint8_t* output, int count, hipStream_t stream)
{
    dim3 block = block_dims(count);
    dim3 grid = grid_dims(count);
    post_process_result_kernel << <grid, block, 0, stream >> > (input, output, count);
}
