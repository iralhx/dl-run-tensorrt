#include "hip/hip_runtime.h"
#include "Segformer_kernel.h"
#include <opencv2/opencv.hpp>


static __global__ void process_imgage_kernel(uint8_t* input, float* output, int allPixel)
{
    int tid = blockDim.x * threadIdx.y + threadIdx.x;
    int blockNum = blockDim.x * blockDim.y;
    int block_offset = blockNum * blockIdx.x;
    int num_a_row = blockNum * gridDim.x;
    int row_offset = num_a_row * blockIdx.y;
    int position = tid + block_offset + row_offset;

    if (position >= allPixel) 
    {
        printf("input  CUOWU \n");

        return;
    }
    int index = position * 3;
    output[index] = input[index] / 255.0f;
    output[index + 1] = input[index + 1] / 255.0f;
    output[index + 2] = input[index + 2] / 255.0f;
    //printf("input %d  ,%d  , %d\n", p1, *p2, *p3);
    //printf("output %f  ,%f  , %f\n", output[index], output[index+1], output[index+2]);
    //uint8_t* p1 = input;  // ָ����������ĵ�һ��Ԫ��
    //printf("First channel value in the input: %d\n", position);
}



void app::process_imgage(uint8_t* input, float* output, int height, int width, hipStream_t stream)
{

    // mask_weights is mask_dim(32 element) gpu pointer
    dim3 grid((width + 31) / 32, (height + 31) / 32);
    dim3 block(32, 32);

    process_imgage_kernel << <grid, block, 0, stream >> > (input, output,height* width);
}




