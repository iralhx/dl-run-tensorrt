#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include"yolov8seg_kernel.h"

//ǰ�ĸ������꣬����ȫ�������Ŷ�,�ں����������
static __global__ void decode_kernel(float* predict, int num_bboxes, int num_classes, int num_mask,
    float confidence_threshold, float* invert_affine_matrix, float* parray, int max_objects) {

    int position = blockDim.x * blockIdx.x + threadIdx.x;
    if (position >= num_bboxes) return;

    float* pitem = predict + (4 + num_classes+ num_mask) * position;

    float* class_confidence = pitem + 4;
    float confidence = *class_confidence++;
    int label = 0;
    for (int i = 1; i < num_classes; ++i, ++class_confidence) {
        if (*class_confidence > confidence) {
            confidence = *class_confidence;
            label = i;
        }
    }

    // confidence *= objectness;
    if (confidence < confidence_threshold)
        return;

    int index = atomicAdd(parray, 1);
    if (index >= max_objects)
        return;
    // printf("index %d max_objects %d\n", index,max_objects);
    float cx = pitem[0];
    float cy = pitem[1];
    float width = pitem[2];
    float height = pitem[3];

    float left = cx - width * 0.5f;
    float top = cy - height * 0.5f;
    float right = cx + width * 0.5f;
    float bottom = cy + height * 0.5f;

    affine_project(invert_affine_matrix, left, top, &left, &top);
    affine_project(invert_affine_matrix, right, bottom, &right, &bottom);


    float* pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
    *pout_item++ = left;
    *pout_item++ = top;
    *pout_item++ = right;
    *pout_item++ = bottom;
    *pout_item++ = confidence;
    *pout_item++ = label;
    *pout_item++ = 1; // 1 = keep, 0 = ignore
    *pout_item++ = position;//������mask��λ��
}

void app::decode_seg_result(float* predict, int num_bboxes, int num_class,int sun_mask, float confidence_threshold, float* invert_affine_matrix, float* parray, int max_objects)
{
    dim3 block = block_dims(num_bboxes);
    dim3 grid = grid_dims(num_bboxes);
    decode_kernel << < grid, block >> > (predict, num_bboxes, num_class, sun_mask,confidence_threshold, invert_affine_matrix, parray, max_objects);
}


static __global__ void decode_single_mask_kernel(int left, int top, float* mask_weights,
    float* mask_predict, int mask_width,
    int mask_height, unsigned char* mask_out,
    int mask_dim, int out_width, int out_height) {
    // mask_predict to mask_out
    // mask_weights @ mask_predict
    int dx = blockDim.x * blockIdx.x + threadIdx.x;
    int dy = blockDim.y * blockIdx.y + threadIdx.y;
    if (dx >= out_width || dy >= out_height) return;

    int sx = left + dx;
    int sy = top + dy;
    if (sx < 0 || sx >= mask_width || sy < 0 || sy >= mask_height) {
        mask_out[dy * out_width + dx] = 0;
        return;
    }

    float cumprod = 0;
    for (int ic = 0; ic < mask_dim; ++ic) {
        float cval = mask_predict[(ic * mask_height + sy) * mask_width + sx];
        float wval = mask_weights[ic];
        cumprod += cval * wval;
    }

    float alpha = 1.0f / (1.0f + exp(-cumprod));
    mask_out[dy * out_width + dx] = alpha * 255;
}

void app::decode_single_mask(float left, float top, float* mask_weights, float* mask_predict,
    int mask_width, int mask_height, unsigned char* mask_out,
    int mask_dim, int out_width, int out_height, hipStream_t stream) {
    // mask_weights is mask_dim(32 element) gpu pointer
    dim3 grid((out_width + 31) / 32, (out_height + 31) / 32);
    dim3 block(32, 32);

    decode_single_mask_kernel<<<grid, block, 0, stream >>>(
        left, top, mask_weights, mask_predict, mask_width, mask_height, mask_out, mask_dim, out_width,
        out_height);
}
