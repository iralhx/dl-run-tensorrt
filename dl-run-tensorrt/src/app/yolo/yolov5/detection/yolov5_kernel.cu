#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include"yolov5_kernel.h"


//ǰ�ĸ������꣬����ȫ�������Ŷ�
static __global__ void decode_kernel(float* predict, int num_bboxes, int num_classes, float confidence_threshold, float* invert_affine_matrix, float* parray, int max_objects) {

    int position = blockDim.x * blockIdx.x + threadIdx.x;
    if (position >= num_bboxes) return;

    float* pitem = predict + (5 + num_classes) * position;

    float* class_confidence = pitem + 5;
    float confidence = *class_confidence++;
    int label = 0;
    for (int i = 1; i < num_classes; ++i, ++class_confidence) {
        if (*class_confidence > confidence) {
            confidence = *class_confidence;
            label = i;

        }

        if (*class_confidence>1)
        {
            printf("*class_confidence :%f  \n", *class_confidence);
        }

    }

    // confidence *= objectness;
    if (confidence < confidence_threshold)
        return;

    int index = atomicAdd(parray, 1);
    if (index >= max_objects)
        return;
    // printf("index %d max_objects %d\n", index,max_objects);
    float cx = pitem[0];
    float cy = pitem[1];
    float width = pitem[2];
    float height = pitem[3];

    float left = cx - width * 0.5f;
    float top = cy - height * 0.5f;
    float right = cx + width * 0.5f;
    float bottom = cy + height * 0.5f;

    affine_project(invert_affine_matrix, left, top, &left, &top);
    affine_project(invert_affine_matrix, right, bottom, &right, &bottom);


    float* pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
    *pout_item++ = left;
    *pout_item++ = top;
    *pout_item++ = right;
    *pout_item++ = bottom;
    *pout_item++ = confidence;
    *pout_item++ = label;
    *pout_item++ = 1; // 1 = keep, 0 = ignore
}

void app::v5_decode_result(float* predict, int num_bboxes, int num_class, float confidence_threshold, float* invert_affine_matrix, float* parray, int max_objects)
{
    dim3 block = block_dims(num_bboxes);
    dim3 grid = grid_dims(num_bboxes);
    decode_kernel << < grid, block >> > (predict, num_bboxes, num_class, confidence_threshold, invert_affine_matrix, parray, max_objects);
}