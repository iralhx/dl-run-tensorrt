#include "hip/hip_runtime.h"
#include "decode_kernel.h"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime.h>
//84*8400
//8400*84
__global__ void transpose_kernel(float *src,int dim1, int dim2,float *dst){

    int position = blockDim.x * blockIdx.x + threadIdx.x;
    if (position >= dim1 * dim2)
    {
        return;
    }
    int dx = position % dim2;
    int dy = position / dim2;
    int detIndex = dx * dim1 + dy;

    *(dst+ detIndex) = src[position];
}


void app::transposeDevice(float* src, int dim1, int dim2, float* dst)
{
    int total = dim1 * dim2;
    dim3 block = block_dims(total);
    dim3 grid = grid_dims(total);

    transpose_kernel << < grid, block >> > (src, dim1, dim2, dst);
}



static __device__ void affine_project(float* matrix, float x, float y, float* ox, float* oy) {
    *ox = matrix[0] * x + matrix[1] * y + matrix[2];
    *oy = matrix[3] * x + matrix[4] * y + matrix[5];
}


const int NUM_BOX_ELEMENT = 7;      // left, top, right, bottom, confidence, class, keepflag
//ǰ�ĸ������꣬����ȫ�������Ŷ�
static __global__ void decode_kernel(float* predict, int num_bboxes, int num_classes, float confidence_threshold, float* invert_affine_matrix, float* parray, int max_objects) {

    int position = blockDim.x * blockIdx.x + threadIdx.x;
    if (position >= num_bboxes) return;

    float* pitem = predict + (4 + num_classes) * position;

    float* class_confidence = pitem + 4;
    float confidence = *class_confidence++;
    int label = 0;
    for (int i = 1; i < num_classes; ++i, ++class_confidence) {
        if (*class_confidence > confidence) {
            confidence = *class_confidence;
            label = i;
        }
    }

    // confidence *= objectness;
    if (confidence < confidence_threshold)
        return;

    int index = atomicAdd(parray, 1);
    if (index >= max_objects)
        return;
    // printf("index %d max_objects %d\n", index,max_objects);
    float cx = pitem[0];
    float cy = pitem[1];
    float width = pitem[2];
    float height = pitem[3];

    float left = cx - width * 0.5f;
    float top = cy - height * 0.5f;
    float right = cx + width * 0.5f;
    float bottom = cy + height * 0.5f;

    affine_project(invert_affine_matrix, left, top, &left, &top);
    affine_project(invert_affine_matrix, right, bottom, &right, &bottom);


    float* pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
    *pout_item++ = left;
    *pout_item++ = top;
    *pout_item++ = right;
    *pout_item++ = bottom;
    *pout_item++ = confidence;
    *pout_item++ = label;
    *pout_item++ = 1; // 1 = keep, 0 = ignore
}

void app::decode_result(float* predict, int num_bboxes, int num_class, float confidence_threshold, float* invert_affine_matrix, float* parray, int max_objects)
{
    dim3 block = block_dims(num_bboxes);
    dim3 grid = grid_dims(num_bboxes);
    decode_kernel << < grid, block >> > (predict, num_bboxes, num_class, confidence_threshold, invert_affine_matrix, parray, max_objects);
}



static __device__ float box_iou(
    float aleft, float atop, float aright, float abottom,
    float bleft, float btop, float bright, float bbottom
) {

    float cleft = max(aleft, bleft);
    float ctop = max(atop, btop);
    float cright = min(aright, bright);
    float cbottom = min(abottom, bbottom);

    float c_area = max(cright - cleft, 0.0f) * max(cbottom - ctop, 0.0f);
    if (c_area == 0.0f)
        return 0.0f;
    float a_area = max(0.0f, aright - aleft) * max(0.0f, abottom - atop);
    float b_area = max(0.0f, bright - bleft) * max(0.0f, bbottom - btop);
    return c_area / (a_area + b_area - c_area);
}

static __global__ void nms_kernel(float* bboxes, int max_objects, float threshold) {

    int position = (blockDim.x * blockIdx.x + threadIdx.x);
    int count = min((int)*bboxes, max_objects);
    if (position >= count)
        return;

    // left, top, right, bottom, confidence, class, keepflag
    float* pcurrent = bboxes + 1 + position * NUM_BOX_ELEMENT;
    for (int i = 0; i < count; ++i) {
        float* pitem = bboxes + 1 + i * NUM_BOX_ELEMENT;
        if (i == position || pcurrent[5] != pitem[5]) continue;

        if (pitem[4] >= pcurrent[4]) {
            if (pitem[4] == pcurrent[4] && i < position)
                continue;

            float iou = box_iou(
                pcurrent[0], pcurrent[1], pcurrent[2], pcurrent[3],
                pitem[0], pitem[1], pitem[2], pitem[3]
            );

            if (iou > threshold) {
                pcurrent[6] = 0;  // 1=keep, 0=ignore
                return;
            }
        }
    }
}

void app::nms_kernel_invoker(float* parray, float nms_threshold, int max_objects) {

    dim3 block = block_dims (max_objects);
    dim3 grid = grid_dims(max_objects) ;
    nms_kernel << <grid, block >> > (parray, max_objects, nms_threshold);
}


__global__ void warpaffine_kernel(
    uint8_t* src, int src_line_size, int src_width,
    int src_height, float* dst, int dst_width,
    int dst_height, uint8_t const_value_st,
    float* d2s, int edge) {
    int position = blockDim.x * blockIdx.x + threadIdx.x;
    if (position >= edge) return;

    float m_x1 = d2s[0];
    float m_y1 = d2s[1];
    float m_z1 = d2s[2];
    float m_x2 = d2s[3];
    float m_y2 = d2s[4];
    float m_z2 = d2s[5];

    int dx = position % dst_width;
    int dy = position / dst_width;
    float src_x = m_x1 * dx + m_y1 * dy + m_z1;
    float src_y = m_x2 * dx + m_y2 * dy + m_z2;
    float c0, c1, c2;


    if (src_x <= -1 || src_x >= src_width || src_y <= -1 || src_y >= src_height) {
        c0 = const_value_st;
        c1 = const_value_st;
        c2 = const_value_st;
    }
    else {
        int x_low = floorf(src_x);
        int y_low = floorf(src_y);
        int x_high = x_low + 1;
        int y_high = y_low + 1;


        float ly = src_y - y_low;
        float lx = src_x - x_low;
        float hy = 1 - ly;
        float hx = 1 - lx;
        float w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
        uint8_t* v1 = src + y_low * src_line_size + x_low * 3;
        uint8_t* v2 = src + y_low * src_line_size + x_high * 3;
        uint8_t* v3 = src + y_high * src_line_size + x_low * 3;
        uint8_t* v4 = src + y_high * src_line_size + x_high * 3;
        c0 = w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0];
        c1 = w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1];
        c2 = w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2];
    }
    // bgr -> rgb
    float temp = c2;
    c2 = c0;
    c0 = temp;

    //// normalization
    c0 /= 255.0f;
    c1 /= 255.0f;
    c2 /= 255.0f;

    // rgbrgbrgb -> rrrgggbbb
    int area = dst_height * dst_width;
    float* pdst_c0 = dst + dy * dst_width + dx;
    float* pdst_c1 = pdst_c0 + area;
    float* pdst_c2 = pdst_c1 + area;
    *pdst_c0 = c0;
    *pdst_c1 = c1;
    *pdst_c2 = c2;
}


void app::preprocess_kernel_img(
    uint8_t* src, int src_width, int src_height,
    float* dst, int dst_width, int dst_height,
    float* d2i, hipStream_t stream) {
    int all = dst_width * dst_height;
    dim3 block = block_dims(all);
    dim3 grid = grid_dims(all);
    warpaffine_kernel << <grid, block, 0, stream >> > (
        src, src_width * 3, src_width,
        src_height, dst, dst_width,
        dst_height, 128, d2i, all);

}
