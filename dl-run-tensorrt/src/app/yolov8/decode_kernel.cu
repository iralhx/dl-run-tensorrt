#include "hip/hip_runtime.h"
#include "decode_kernel.h"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime.h>
//1*84*8400
//1*8400*84
__global__ void transpose_kernel(float *src,int num_bboxes, int num_class,float *dst){

    int weidthIndex = blockDim.x * blockIdx.x + threadIdx.x;
    int heightIndex = blockDim.y * blockIdx.y + threadIdx.y;
    if (weidthIndex>=num_bboxes||heightIndex>=(num_class+4))
    {
        return;
    }

    int globalIndex = heightIndex * num_bboxes+weidthIndex;
    int detIndex = weidthIndex * (num_class + 4) + heightIndex;
    *(dst+ detIndex) = src[globalIndex];
}


void app::transposeDevice(float* src, int num_bboxes, int num_class, float* dst)
{
    dim3 grid_size(ceil(num_bboxes  / BLOCK_SIZE),
        ceil((num_class+4)/ BLOCK_SIZE));
    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);

    transpose_kernel << < grid_size, block_size >> > (src, num_bboxes, num_class, dst);
}





const int NUM_BOX_ELEMENT = 7;      // left, top, right, bottom, confidence, class, keepflag
//ǰ�ĸ������꣬����ȫ�������Ŷ�
static __global__ void decode_kernel(float* predict, int num_bboxes, int num_classes, float confidence_threshold, float* parray, int max_objects) {

    int one_block_all = blockDim.x * blockDim.y;
    int tid = blockDim.x * threadIdx.y + threadIdx.x;
    int grid_offset = one_block_all * gridDim.x * blockIdx.y + one_block_all * blockIdx.x;
    int index = grid_offset + tid;

    if (index >= num_bboxes) {
        return;
    }

    float* pitem = predict + (4 + num_classes) * index*sizeof(float);

    float* class_confidence = pitem + 4;
    float confidence = *class_confidence++;
    int label = 0;
    for (int i = 1; i < num_classes; ++i, ++class_confidence) {
        if (*class_confidence > confidence) {
            confidence = *class_confidence;
            label = i;
        }
    }

    if (confidence < confidence_threshold)
        return;

    int indexResult = atomicAdd(parray, 1);
    if (indexResult >= max_objects)
        return;

    float cx = *pitem++;
    float cy = *pitem++;
    float width = *pitem++;
    float height = *pitem++;
    float left = cx - width * 0.5f;
    float top = cy - height * 0.5f;
    float right = cx + width * 0.5f;
    float bottom = cy + height * 0.5f;
    printf("Source,cx:%f,cy:%f,width:%f,height:%f\n",
        cx, cy, width, height);

    float* pout_item = parray + 1 + indexResult * NUM_BOX_ELEMENT;
    *pout_item++ = left;
    *pout_item++ = top;
    *pout_item++ = right;
    *pout_item++ = bottom;
    *pout_item++ = confidence;
    *pout_item++ = label;
    *pout_item++ = 1; // 1 = keep, 0 = ignore
    printf("���ǵ�:%d,left:%f,top:%f,right:%f,bottom:%f,confidence:%f,label:%f\n", 
        indexResult , left, top, right,bottom,confidence,label);

};



void app::decode_result(float* predict, int num_bboxes, int num_class, float confidence_threshold, float* parray, int max_objects)
{
    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid_size( ceil((num_class + 4) / BLOCK_SIZE),
        ceil(num_bboxes / BLOCK_SIZE));
    decode_kernel << < grid_size, block_size >> > (predict, num_bboxes, num_class, confidence_threshold, parray, max_objects);
}



static __device__ float box_iou(
    float aleft, float atop, float aright, float abottom,
    float bleft, float btop, float bright, float bbottom
) {

    float cleft = max(aleft, bleft);
    float ctop = max(atop, btop);
    float cright = min(aright, bright);
    float cbottom = min(abottom, bbottom);

    float c_area = max(cright - cleft, 0.0f) * max(cbottom - ctop, 0.0f);
    if (c_area == 0.0f)
        return 0.0f;

    float a_area = max(0.0f, aright - aleft) * max(0.0f, abottom - atop);
    float b_area = max(0.0f, bright - bleft) * max(0.0f, bbottom - btop);
    return c_area / (a_area + b_area - c_area);
}

static __global__ void nms_kernel(float* bboxes, int max_objects, float threshold) {

    int position = (blockDim.x * blockIdx.x + threadIdx.x);
    int count = min((int)*bboxes, max_objects);
    if (position >= count)
        return;

    // left, top, right, bottom, confidence, class, keepflag
    float* pcurrent = bboxes + 1 + position * NUM_BOX_ELEMENT;
    for (int i = 0; i < count; ++i) {
        float* pitem = bboxes + 1 + i * NUM_BOX_ELEMENT;
        if (i == position || pcurrent[5] != pitem[5]) continue;

        if (pitem[4] >= pcurrent[4]) {
            if (pitem[4] == pcurrent[4] && i < position)
                continue;

            float iou = box_iou(
                pcurrent[0], pcurrent[1], pcurrent[2], pcurrent[3],
                pitem[0], pitem[1], pitem[2], pitem[3]
            );

            if (iou > threshold) {
                pcurrent[6] = 0;  // 1=keep, 0=ignore
                return;
            }
        }
    }
}

void app::nms_kernel_invoker(float* parray, float nms_threshold, int max_objects) {

    dim3 block = block_dims (max_objects);
    dim3 grid = grid_dims(max_objects) ;
    nms_kernel << <grid, block >> > (parray, max_objects, nms_threshold);
}


