#include "hip/hip_runtime.h"
#include "decode_kernel.h"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime.h>
//1*84*8400
//1*8400*84
__global__ void transpose_kernel(float *src,int num_bboxes, int num_class,float *dst){

    int weidthIndex = blockDim.x * blockIdx.x + threadIdx.x;
    int heightIndex = blockDim.y * blockIdx.y + threadIdx.y;
    if (weidthIndex>=num_bboxes||heightIndex>=(num_class+4))
    {
        return;
    }

    int globalIndex = heightIndex * num_bboxes+weidthIndex;
    int detIndex = weidthIndex * (num_class + 4) + heightIndex;
    *(dst+ detIndex) = src[globalIndex];
}


void app::transposeDevice(float* src, int num_bboxes, int num_class, float* dst)
{
    dim3 grid_size(ceil(num_bboxes  / BLOCK_SIZE),
        ceil((num_class+4)/ BLOCK_SIZE));
    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);

    transpose_kernel << < grid_size, block_size >> > (src, num_bboxes, num_class, dst);
}



static __device__ void affine_project(float* matrix, float x, float y, float* ox, float* oy) {
    *ox = matrix[0] * x + matrix[1] * y + matrix[2];
    *oy = matrix[3] * x + matrix[4] * y + matrix[5];
}


const int NUM_BOX_ELEMENT = 7;      // left, top, right, bottom, confidence, class, keepflag
//ǰ�ĸ������꣬����ȫ�������Ŷ�
static __global__ void decode_kernel(float* predict, int num_bboxes, int num_classes, float confidence_threshold, float* invert_affine_matrix, float* parray, int max_objects) {

    int position = blockDim.x * blockIdx.x + threadIdx.x;
    if (position >= num_bboxes) return;

    float* pitem = predict + (4 + num_classes) * position;
    // float objectness = pitem[4];
    // if(objectness < confidence_threshold)
    //     return;

    float* class_confidence = pitem + 4;
    float confidence = *class_confidence++;
    int label = 0;
    for (int i = 1; i < num_classes; ++i, ++class_confidence) {
        if (*class_confidence > confidence) {
            confidence = *class_confidence;
            label = i;
        }
    }

    // confidence *= objectness;
    if (confidence < confidence_threshold)
        return;

    int index = atomicAdd(parray, 1);
    if (index >= max_objects)
        return;
    // printf("index %d max_objects %d\n", index,max_objects);
    float cx = pitem[0];
    float cy = pitem[1];
    float width = pitem[2];
    float height = pitem[3];

    float left = cx - width * 0.5f;
    float top = cy - height * 0.5f;
    float right = cx + width * 0.5f;
    float bottom = cy + height * 0.5f;

    affine_project(invert_affine_matrix, left, top, &left, &top);
    affine_project(invert_affine_matrix, right, bottom, &right, &bottom);


    float* pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
    *pout_item++ = left;
    *pout_item++ = top;
    *pout_item++ = right;
    *pout_item++ = bottom;
    *pout_item++ = confidence;
    *pout_item++ = label;
    *pout_item++ = 1; // 1 = keep, 0 = ignore
}

void app::decode_result(float* predict, int num_bboxes, int num_class, float confidence_threshold, float* invert_affine_matrix, float* parray, int max_objects)
{
    dim3 block = block_dims(max_objects);
    dim3 grid = grid_dims(max_objects);
    decode_kernel << < grid, block >> > (predict, num_bboxes, num_class, confidence_threshold, invert_affine_matrix, parray, max_objects);
}



static __device__ float box_iou(
    float aleft, float atop, float aright, float abottom,
    float bleft, float btop, float bright, float bbottom
) {

    float cleft = max(aleft, bleft);
    float ctop = max(atop, btop);
    float cright = min(aright, bright);
    float cbottom = min(abottom, bbottom);

    float c_area = max(cright - cleft, 0.0f) * max(cbottom - ctop, 0.0f);
    if (c_area == 0.0f)
        return 0.0f;

    float a_area = max(0.0f, aright - aleft) * max(0.0f, abottom - atop);
    float b_area = max(0.0f, bright - bleft) * max(0.0f, bbottom - btop);
    return c_area / (a_area + b_area - c_area);
}

static __global__ void nms_kernel(float* bboxes, int max_objects, float threshold) {

    int position = (blockDim.x * blockIdx.x + threadIdx.x);
    int count = min((int)*bboxes, max_objects);
    if (position >= count)
        return;

    // left, top, right, bottom, confidence, class, keepflag
    float* pcurrent = bboxes + 1 + position * NUM_BOX_ELEMENT;
    for (int i = 0; i < count; ++i) {
        float* pitem = bboxes + 1 + i * NUM_BOX_ELEMENT;
        if (i == position || pcurrent[5] != pitem[5]) continue;

        if (pitem[4] >= pcurrent[4]) {
            if (pitem[4] == pcurrent[4] && i < position)
                continue;

            float iou = box_iou(
                pcurrent[0], pcurrent[1], pcurrent[2], pcurrent[3],
                pitem[0], pitem[1], pitem[2], pitem[3]
            );

            if (iou > threshold) {
                pcurrent[6] = 0;  // 1=keep, 0=ignore
                return;
            }
        }
    }
}

void app::nms_kernel_invoker(float* parray, float nms_threshold, int max_objects) {

    dim3 block = block_dims (max_objects);
    dim3 grid = grid_dims(max_objects) ;
    nms_kernel << <grid, block >> > (parray, max_objects, nms_threshold);
}




__global__ void warpaffine_kernel(
    float* src, int src_line_size, int src_width,
    int src_height, float* dst, int dst_width,
    int dst_height, uint8_t const_value_st,
    float* d2i, int edge) {
    int position = blockDim.x * blockIdx.x + threadIdx.x;
    if (position >= edge) return;

    float m_x1 = d2i[0];
    float m_y1 = d2i[1];
    float m_z1 = d2i[2];
    float m_x2 = d2i[3];
    float m_y2 = d2i[4];
    float m_z2 = d2i[5];

    int dx = position % dst_width;
    int dy = position / dst_width;
    float src_x = m_x1 * dx + m_y1 * dy + m_z1 + 0.5f;
    float src_y = m_x2 * dx + m_y2 * dy + m_z2 + 0.5f;
    float c0, c1, c2;

    if (src_x <= -1 || src_x >= src_width || src_y <= -1 || src_y >= src_height) {
        // out of range
        c0 = const_value_st;
        c1 = const_value_st;
        c2 = const_value_st;
    }
    else {
        //˫���Բ�ֵ
        //����ȡ��
        int y_low = floorf(src_y);
        int x_low = floorf(src_x);
        int y_high = y_low + 1;
        int x_high = x_low + 1;

        float const_value[] = { const_value_st, const_value_st, const_value_st };
        float ly = src_y - y_low;
        float lx = src_x - x_low;
        float hy = 1 - ly;
        float hx = 1 - lx;
        float w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
        float* v1 = const_value;
        float* v2 = const_value;
        float* v3 = const_value;
        float* v4 = const_value;

        if (y_low >= 0) {
            if (x_low >= 0)
                v1 = src + y_low * src_line_size + x_low * 3;

            if (x_high < src_width)
                v2 = src + y_low * src_line_size + x_high * 3;
        }

        if (y_high < src_height) {
            if (x_low >= 0)
                v3 = src + y_high * src_line_size + x_low * 3;

            if (x_high < src_width)
                v4 = src + y_high * src_line_size + x_high * 3;
        }

        c0 = w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0];
        c1 = w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1];
        c2 = w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2];
    }

    //bgr to rgb 
    float t = c2;
    c2 = c0;
    c0 = t;

    //normalization
    c0 = c0 / 255.0f;
    c1 = c1 / 255.0f;
    c2 = c2 / 255.0f;

    //rgbrgbrgb to rrrgggbbb
    int area = dst_width * dst_height;
    float* pdst_c0 = dst + dy * dst_width + dx;
    float* pdst_c1 = pdst_c0 + area;
    float* pdst_c2 = pdst_c1 + area;
    printf("c0:%d, c1:%f, c2:%f\n", position, c1, c2);
    *pdst_c0 = c0;
    *pdst_c1 = c1;
    *pdst_c2 = c2;
}


void app::preprocess_kernel_img(
    float* src, int src_width, int src_height,
    float* dst, int dst_width, int dst_height,
    float* d2i, hipStream_t stream) {
    int all = dst_width * dst_height;
    dim3 block = block_dims(all);
    dim3 grid = grid_dims(all);
    warpaffine_kernel << <grid, block, 0, stream >> > (
        src, src_width * 3, src_width,
        src_height, dst, dst_width,
        dst_height, 128, d2i, all);

}
