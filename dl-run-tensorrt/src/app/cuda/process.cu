#include "hip/hip_runtime.h"
#include <hip/driver_types.h>
#include <hip/hip_runtime.h>
//#include<crt/math_functions.h>
const int NUM_BOX_ELEMENT = 7;

void transpose_kernel_invoker(float *src,int num_bboxes,int num_elements,float *dst,hipStream_t stream)
{
    int edge = num_bboxes*num_elements;
    int block =256;
    int gird = ceil(edge/(float)block);
    transpose_kernel<<<gird,block,0,stream>>>(src,num_bboxes,num_elements,dst,edge);
}

static __global__ void transpose_kernel(float *src,int num_bboxes, int num_elements,float *dst,int edge)
{
    int position = blockDim.x * blockIdx.x + threadIdx.x;
    if (position>=edge)
        return;
    dst[position]=src[position/num_elements+(position%num_elements)*num_bboxes];

}


static __global__ void decode_kernel(float* predict, int num_bboxes, int num_classes, float confidence_threshold, float* parray, int max_objects){  

    int position = blockDim.x * blockIdx.x + threadIdx.x;
	if (position >= num_bboxes) return;

    float* pitem     = predict + (4 + num_classes) * position;
    // float objectness = pitem[4];
    // if(objectness < confidence_threshold)
    //     return;

    float* class_confidence = pitem + 4;
    float confidence        = *class_confidence++;
    int label               = 0;
    for(int i = 1; i < num_classes; ++i, ++class_confidence){
        if(*class_confidence > confidence){
            confidence = *class_confidence;
            label      = i;
        }
    }

    // confidence *= objectness;
    if(confidence < confidence_threshold)
        return;
   
    int index = atomicAdd(parray, 1);
    if(index >= max_objects)
        return;
    // printf("index %d max_objects %d\n", index,max_objects);
    float cx         = pitem[0];
    float cy         = pitem[1];
    float width      = pitem[2];
    float height     = pitem[3];

    float left   = cx - width * 0.5f;
    float top    = cy - height * 0.5f;
    float right  = cx + width * 0.5f;
    float bottom = cy + height * 0.5f;


    float* pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
    *pout_item++ = left;
    *pout_item++ = top;
    *pout_item++ = right;
    *pout_item++ = bottom;
    *pout_item++ = confidence;
    *pout_item++ = label;
    *pout_item++ = 1; // 1 = keep, 0 = ignore
}

void decode_kernel_invoker(float* predict, int num_bboxes, int num_classes, float confidence_threshold, float* parray, int max_objects, hipStream_t stream)
{
    int block = 256;
    int  grid =  ceil(num_bboxes / (float)block);
        
    decode_kernel<<<grid, block, 0, stream>>>(predict, num_bboxes, num_classes, confidence_threshold, parray, max_objects);
}

static __global__ void nms_kernel(float* bboxes, int max_objects, float threshold){
    int position = (blockDim.x * blockIdx.x + threadIdx.x);
    int count = min((int)*bboxes, max_objects);
    if (position >= count) 
        return;
        
    // left, top, right, bottom, confidence, class, keepflag
    float* pcurrent = bboxes + 1 + position * NUM_BOX_ELEMENT;
    for(int i = 0; i < count; ++i){
        float* pitem = bboxes + 1 + i * NUM_BOX_ELEMENT;
        if(i == position || pcurrent[5] != pitem[5]) continue;

        if(pitem[4] >= pcurrent[4]){
            if(pitem[4] == pcurrent[4] && i < position)
                continue;

            float iou = box_iou(
                pcurrent[0], pcurrent[1], pcurrent[2], pcurrent[3],
                pitem[0],    pitem[1],    pitem[2],    pitem[3]
            );

            if(iou > threshold){
                pcurrent[6] = 0;  // 1=keep, 0=ignore
                return;
            }
        }
    }
} 

void nms_kernel_invoker(float* parray, float nms_threshold, int max_objects, hipStream_t stream){
    int block = max_objects<256? max_objects:256;
    int grid = ceil(max_objects / (float)block);
    nms_kernel<<<grid, block, 0, stream>>>(parray, max_objects, nms_threshold);
}